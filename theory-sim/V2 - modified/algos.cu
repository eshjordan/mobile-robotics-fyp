#include "hip/hip_runtime.h"
/*
Full CUDA translation of algos.py for GPU
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "math.h"
#include "stdio.h"

#include "limits.h"

// #include "perms.cpp"

#define PI 3.14159265358979323846f

#define min(X, Y) (((X) > (Y)) ? (Y) : (X))
#define max(X, Y) (((X) < (Y)) ? (Y) : (X))

#define MIN_NUM_AGENTS 5
#define MAX_NUM_AGENTS 8
#define NUM_ITERS 3

#define THREAD_PER_BLOCK 1024
#define SPLIT_SIZE (2048*1024)

__host__ __device__
float mod(float x, float y) {
    float res = fmodf(x,y);
    if (res < 0) {
        res += y;
    }
    return res;
}

// #define NUM_ITERS 1




// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }




class Agent {
public:
    float theta_u;
    float theta_l;
    int n;
    float eps;

    __host__ __device__
        Agent(float theta_l = NULL, float theta_u = NULL, int n = NULL, float eps = 0)
        : theta_l(theta_l), theta_u(theta_u), n(n), eps(eps) {}

    __host__ __device__
        float mid(float t1, float t2) const {
        if (t2 < t1) t2 += 2 * PI;
        return mod(0.5f * (t1 + t2), 2 * PI);
    }

    __host__ __device__
        float theta_c() const {
        return mid(theta_l, theta_u);
    }

    __host__ __device__
        void update_knowledge(int new_n) {
        n = new_n;
        move_to(theta_c()); // update bounds
    }

    __host__ __device__
        void move_to(float c) {
        theta_l = mod(c - PI / n, 2 * PI);
        theta_u = mod(c + PI / n, 2 * PI);
    }
};



__device__
float d(float t1, float t2) {
    float m = fabsf(t2 - t1);
    return (m < PI) ? m : (2 * PI - m);
}

__device__
float compute_overlap_neg(Agent* i, Agent* j) {
    // allow negative overlap, to compute max gap
    return fminf(PI / i->n + PI / j->n + i->eps / 2 + j->eps / 2 - d(i->theta_c(), j->theta_c()), fminf(2 * PI / i->n, 2 * PI / j->n));
}

__host__ __device__
unsigned long long factorial(int N) {
    int fact = 1, i;
    for (i = 1; i <= N; i++) { fact *= i; }
    return fact;
}

__host__ __device__
unsigned long long power(unsigned long long X, int Y) {
    // compute X^Y
    unsigned long long pwr = 1, i;
    for (i = 0; i < Y; i++) { pwr *= X; }
    return pwr;
}

// Function to generate the nth permutation of the array
__host__ __device__
void getNthPermutation(unsigned long long n, int result[], int num_agents, bool* taken) {
    // Adjust n to be zero-based
    // n -= 1;

    // clear arrays
    for (int i = 0; i < num_agents; i++) {
        result[i] = -1;
        taken[i] = false;
    }

    // Compute the permutation
    for (int i = 0; i < num_agents; i++) {
        int fact = factorial(num_agents - 1 - i);
        int index = n / fact;
        int count = 0;

        // Find the (index + 1)th available number
        for (int j = 0; j < num_agents; j++) {
            if (!taken[j]) {
                if (count == index) {
                    //result[i] = nums[j];
                    result[i] = j;
                    taken[j] = true;
                    break;
                }
                count++;
            }
        }

        n %= fact;
    }
}


__host__ __device__
void getNthPermutationMultIters(int num_iters, unsigned long long n, int result[], int num_agents, bool* taken) {


    unsigned long long NUM_TASKS = factorial(num_agents); // for a single iter

    for (int i = 0; i < num_iters; i++) {
        unsigned long long base = power(NUM_TASKS, i);
        getNthPermutation((n / base) % NUM_TASKS, &(result[i * num_agents]), num_agents, taken);
    }
}


__device__
int compute_nn(const Agent* agents, int k, int NUM_AGENTS) {
    float current_min = 2 * PI;
    int current_j = -1;
    for (int j = 0; j < NUM_AGENTS; ++j) {
        if (j == k) continue;
        float dist = mod(agents[k].theta_c() - agents[j].theta_c(), 2 * PI);
        if (dist < current_min) {
            current_min = dist;
            current_j = j;
        }
    }
    return current_j;
}


__host__
void sequential_add_in(int num_agents, Agent* agents) {

    /*
     * In: num_agents
     * Out: agents (space already allocated)
    */

    Agent new_agent;
    new_agent.update_knowledge(2);
    new_agent.move_to(0);

    agents[0] = Agent();
    agents[0].update_knowledge(2);
    agents[0].move_to(0);

    agents[1] = Agent();
    agents[1].update_knowledge(3);
    agents[1].move_to(2 * PI / 3);

    for (int p = 2; p < num_agents - 1; p++) {

        float theta_c = PI;
        for (int m = 3; m <= p + 1; m++) {
            theta_c += PI / m;
        }
        theta_c -= PI / (p + 2);

        agents[p] = Agent();
        agents[p].update_knowledge(p + 2);
        agents[p].move_to(theta_c);
    }

    // last one
    float theta_c = PI;
    for (int m = 3; m <= num_agents; m++) {
        theta_c += PI / m;
    }
    agents[num_agents - 1] = Agent();
    agents[num_agents - 1].update_knowledge(num_agents);
    agents[num_agents - 1].move_to(theta_c);
}


__global__
void pairwise_interaction(int NUM_AGENTS, float* max_gap_arr, Agent* d_agents, Agent* d_agents_init, int* d_krange, bool* d_taken, int split, int NUM_TASKS) {

    //unsigned long long NUM_TASKS, TASK_IDX;
    //NUM_TASKS = factorial(NUM_AGENTS);
    unsigned long long TASK_IDX;
    TASK_IDX = THREAD_PER_BLOCK * blockIdx.x + threadIdx.x;

    // don't compute if outside task range
    if (TASK_IDX < NUM_TASKS) {

        int KRANGE_SIZE = NUM_AGENTS * NUM_ITERS;

        Agent* agents = &(d_agents[TASK_IDX * NUM_AGENTS]);
        int* krange = &(d_krange[TASK_IDX * NUM_AGENTS * NUM_ITERS]);
        bool* taken = &(d_taken[TASK_IDX * NUM_AGENTS]);

        float max_gap = 0, overlap_neg;
        int i = 0, k, nn, new_n;
        Agent* agent_k, * agent_k_nn; // *agents;

        // init agents
        //hipMemcpy(agents, d_agents_init, NUM_AGENTS * sizeof(Agent), cudaMemcpyDevicetoDevice);
        for (int i = 0; i < NUM_AGENTS; i++) {
            agents[i] = d_agents_init[i];
        }

        // sequential_add_in(NUM_AGENTS, agents);
        //getNthPermutation(TASK_IDX, krange, NUM_AGENTS, taken);
        getNthPermutationMultIters(NUM_ITERS, (split*SPLIT_SIZE) + TASK_IDX, krange, NUM_AGENTS, taken);

        while (i < KRANGE_SIZE) {
            k = krange[i];
            agent_k = &agents[k];
            nn = compute_nn(agents, k, NUM_AGENTS);
            agent_k_nn = &agents[nn];
            overlap_neg = compute_overlap_neg(agent_k, agent_k_nn);
            if (-overlap_neg > max_gap) {
                max_gap = -overlap_neg;
            }

            // Assume run_flag not unset

            new_n = max(agent_k->n, agent_k_nn->n);
            agent_k->update_knowledge(new_n);
            agent_k_nn->update_knowledge(new_n);
            agent_k->move_to(agent_k_nn->theta_c() + 2 * PI / new_n);
            i += 1;
        }
    
        max_gap_arr[TASK_IDX] = max_gap;
        //max_gap_arr[TASK_IDX] = TASK_IDX;
    }

}




int main() {

    printf("Hello\n\n");

    float* max_gap_arr;
    float max_gap;
    int NUM_BLOCKS, NUM_THREADS, split;
    unsigned long long NUM_TASKS, TOT_NUM_TASKS, arg_max_gap;

    Agent* agents_init, * d_agents_init, * d_agents;
    int* d_krange;
    bool* d_taken;

    for (int NUM_AGENTS = MIN_NUM_AGENTS; NUM_AGENTS <= MAX_NUM_AGENTS; NUM_AGENTS++) {


        //TOT_NUM_TASKS = factorial(NUM_AGENTS);
        NUM_TASKS = factorial(NUM_AGENTS);
        if (NUM_TASKS > pow(ULLONG_MAX, 1.0 / NUM_ITERS)) {
            printf("%i agents with %i iterations is too large. Stopping.\n", NUM_AGENTS, NUM_ITERS);
            break;
        }
        TOT_NUM_TASKS = power(NUM_TASKS, NUM_ITERS);

        printf("Computing max gap for %i agents with %i iterations: %llu tasks\n", NUM_AGENTS, NUM_ITERS, TOT_NUM_TASKS);


        // Allocate memory

        // Max gap array
        hipMallocManaged(&max_gap_arr, SPLIT_SIZE * sizeof(float));
        // Agents init array
        agents_init = (Agent*)malloc(NUM_AGENTS * sizeof(Agent));
        sequential_add_in(NUM_AGENTS, agents_init);
        hipMalloc(&d_agents_init, NUM_AGENTS * sizeof(Agent));
        hipMemcpy(d_agents_init, agents_init, NUM_AGENTS * sizeof(Agent), hipMemcpyHostToDevice);
        // Agents array
        hipMalloc(&d_agents, SPLIT_SIZE * NUM_AGENTS * sizeof(Agent));
        // d_krange
        hipMalloc(&d_krange, SPLIT_SIZE * NUM_AGENTS * NUM_ITERS * sizeof(int)); // agents will compute their own krange (???????)
        // taken
        hipMalloc(&d_taken, SPLIT_SIZE * NUM_AGENTS * sizeof(bool)); // each task only needs 1xNUM_AGENTS array of bools for 'taken'


        // Split up into SPLIT_SIZE
        split = 0;
        max_gap = 0;
        arg_max_gap = 0;
        while (TOT_NUM_TASKS > 0)
        {
            NUM_TASKS = min(TOT_NUM_TASKS, SPLIT_SIZE);

            //// Reset stuff
            //for (int i = 0; i < NUM_TASKS; i++) {
            //    hipMemcpy(&(d_agents[i * NUM_AGENTS]), agents_init, NUM_AGENTS * sizeof(Agent), hipMemcpyHostToDevice);
            //}


            //printf("Max gap array initialised\n");

            NUM_BLOCKS = 1 + ((NUM_TASKS-1) / THREAD_PER_BLOCK);
            //NUM_THREADS = min(THREAD_PER_BLOCK, NUM_TASKS);
            NUM_THREADS = THREAD_PER_BLOCK;

            printf("\rUsing %i blocks and %i threads per block...", NUM_BLOCKS, NUM_THREADS);

            pairwise_interaction << <NUM_BLOCKS, NUM_THREADS >> > (NUM_AGENTS, max_gap_arr, d_agents, d_agents_init, d_krange, d_taken, split, NUM_TASKS);
            hipDeviceSynchronize();



            //printf("Kernel completed\n");

            // find max
            //printf("Computing max gap\n");
            for (int i = 0; i < NUM_TASKS; i++) {
                //printf("%.3f ", max_gap_arr[i]);
                if (max_gap_arr[i] > max_gap) {
                    max_gap = max_gap_arr[i];
                    arg_max_gap = (split*SPLIT_SIZE) + i;
                    //printf("Max gap is now %.3f\n", max_gap);
                }
            }

            printf("%llu done", (split * SPLIT_SIZE) + NUM_TASKS);

            TOT_NUM_TASKS -= NUM_TASKS;
            split += 1;

            
        }


        //printf("\nComputed max gap\n");

        printf("\nMax gap for %i agents over %i iterations is %.4f\n", NUM_AGENTS, NUM_ITERS, max_gap);
        int* arg_max_gap_perm = (int*)malloc(NUM_AGENTS * NUM_ITERS * sizeof(int));
        bool* taken = (bool*)malloc(NUM_AGENTS * sizeof(bool));
        for (int i = 0; i < NUM_AGENTS; i++) {
            taken[i] = false;
        }
        getNthPermutationMultIters(NUM_ITERS, arg_max_gap, arg_max_gap_perm, NUM_AGENTS, taken);
        printf("[");
        for (int i = 0; i < NUM_AGENTS * NUM_ITERS; i++) {
            printf(" %i,", arg_max_gap_perm[i]);
        }
        printf("]\n\n");
        free(arg_max_gap_perm);
        free(taken);


        hipFree(max_gap_arr);
        hipFree(d_agents);
        hipFree(d_krange);
        hipFree(d_taken);

        free(agents_init);
    }

    //hipFree(&d_init_arr);

    return 0;
}

